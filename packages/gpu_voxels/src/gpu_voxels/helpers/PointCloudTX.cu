// this is for emacs file handling -*- mode: c++; indent-tabs-mode: nil -*-

// -- BEGIN LICENSE BLOCK ----------------------------------------------
// This file is part of the GPU Voxels Software Library.
//
// This program is free software licensed under the CDDL
// (COMMON DEVELOPMENT AND DISTRIBUTION LICENSE Version 1.0).
// You can find a copy of this license in LICENSE.txt in the top
// directory of the source code.
//
// © Copyright 2014 FZI Forschungszentrum Informatik, Karlsruhe, Germany
//
// -- END LICENSE BLOCK ------------------------------------------------

//----------------------------------------------------------------------
/*!\file
 *
 * \author  Andreas Hermann
 * \date    2018-03-12
 *
 */
//----------------------------------------------------------------------
#include "PointCloudTX.h"
#include <gpu_voxels/helpers/kernels/MetaPointCloudOperations.h>
#include <gpu_voxels/logging/logging_gpu_voxels.h>
#include <gpu_voxels/helpers/kernels/HelperOperations.h>
#include <gpu_voxels/helpers/PointcloudFileHandler.h>

namespace gpu_voxels
{


PointCloudTX::PointCloudTX()
{

}



PointCloudTX::~PointCloudTX()
{

}





void PointCloudTX::transform(const Matrix4f *transform, Vector3f *input_cloud, Vector3f *transformed_cloud, size_t num_points)
{


  Vector3f *transformed_cloud_dev, *input_cloud_dev;
  Matrix4f *m_transformation_dev;
  // Get device pointer from host memory. No allocation or memcpy
  hipHostGetDevicePointer((void **)&input_cloud_dev, (void *) input_cloud , 0);
  hipHostGetDevicePointer((void **)&transformed_cloud_dev, (void *) transformed_cloud, 0);
  hipHostGetDevicePointer((void **)&m_transformation_dev, (void *) transform, 0);


  computeLinearLoad(num_points, &m_blocks, &m_threads_per_block);
  // transform the cloud via Kernel.
  kernelTransformCloud<<< m_blocks, m_threads_per_block >>>
                          (m_transformation_dev,
                           input_cloud_dev,
                           transformed_cloud_dev,
                           num_points);
  CHECK_CUDA_ERROR();

  HANDLE_CUDA_ERROR(hipDeviceSynchronize());
}


}// end namespace gpu_voxels
